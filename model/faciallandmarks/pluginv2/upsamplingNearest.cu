
#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

template<typename T>
__global__ void nearest_neighbor_upsampling( T * __restrict__ odata, const T * __restrict__ idata, const size_t nrElementsIn, const size_t upsamplingFactor, const size_t columnsIn)
{
    // Reference value for Deconvolution layer from TensorRT 4.0 RC (1x32x5x5): 0.2459 ms (TITAN X Pascal) | 0.2175 ms (GTX 1050)
    // Measured value: 0.0036823 ms (TITAN X Pascal, 67x) | 0.0037416 ms (GTX 1050, 58x)

    // Reference value for Deconvolution layer from TensorRT 4.0 RC (1x64x40x40): 0.4772 ms (TITAN X Pascal) | 0.5669 ms (GTX 1050)
    // Measured value: 0.00838435 ms (TITAN X Pascal, 57x) | 0.032248 ms (GTX 1050, 18x)

    // One thread reads in one original value and writes it out to the dedicated output locations
    const int idxIn = blockDim.x * blockIdx.x + threadIdx.x;
    if(idxIn >= nrElementsIn)
        return;
    const int rowIn = idxIn / columnsIn;
    const int colIn = idxIn % columnsIn;
    const int startIdxOut = (rowIn * columnsIn * upsamplingFactor + colIn) * upsamplingFactor;
    const T input_copy = idata[idxIn];
    for(int r = 0; r < upsamplingFactor; r++)
    {
        const int rowIdxOut = startIdxOut + r * columnsIn * upsamplingFactor;
        for(int c = 0; c < upsamplingFactor; c++)
        {
            const int idx_out = rowIdxOut + c;
            odata[idx_out] = input_copy;
        }
    }
}

void deconv_nearest_neighbor_upsampling(hipStream_t stream, void *d_output, const void *d_input,
    int N, int C, int H, int W, int upsamplingFactor, bool is_fp16)
{
    const int sizeInput = N * C * H * W;
    const int threadsPerBlock = 1024;
    const int blocksInGrid = (sizeInput + threadsPerBlock - 1) / threadsPerBlock;
    if (is_fp16)
    {
        nearest_neighbor_upsampling<half><<<blocksInGrid, threadsPerBlock, 0, stream >>>((half *)d_output, (const half *)d_input, sizeInput, upsamplingFactor, W);
    }
    else
    {
        nearest_neighbor_upsampling<float><<<blocksInGrid, threadsPerBlock, 0, stream >>>((float *)d_output, (const float *)d_input, sizeInput, upsamplingFactor, W);
    }
    hipGetLastError();
}
