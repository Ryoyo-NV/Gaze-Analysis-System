#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <assert.h>
#include <iostream>
#include <stdio.h>

#define g_H_Fl 80
#define g_W_Fl 80

template <int H, int W> __global__ void softargmax_kernel_half(const __half2 *srcData, 
   __half *dstDataX,
   __half *dstDataY,
   __half *dstDataProb,
   const int HW,
   const int CHW,
   __half beta,
   __half epsilon)
{
    // Shared memory for storing the maxima per row and after reduction per channel (elementary max operation only available for float, fmaxf):
    __shared__ float s_rowMaximum[H];
    __shared__ float s_channelMaximum;
    // Shared memory for storing the intermediate sums per row and per column, and for saving the softmax denominator (as reciprocate, as __half2):
    __shared__ __half s_sumRows[H];
    __shared__ __half s_sumCols[H];
    __shared__ __half2 s_softmaxDenominatorRcp;

    // We will perform all operations except for fmaxf in half2 mode and will hence convert the constant beta to the __half2 data type:
    __half2 betaHalf2 = __half2half2(beta);

    // Instead of fetching input data from global memory twice, we will use a local buffer per thread:
    __half2 dataLocal[W/2];

    // Information we can infer from this thread:
    const int batch = blockIdx.x;
    const int channel = blockIdx.y;
    const int row = threadIdx.x;

    // This is the starting index of the input elements in __half2 strides (hence the division by two)
    const int elemOffset = (batch * CHW + channel * HW + row * W)/2;

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // PHASE 1: Calculate the maximum per channel for "accurate" Softmax mode.
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
    // 1a) Find maximum per row:

    float maxElem = -FLT_MAX;
    // Using one thread, loop over all columns:
#pragma unroll
    for (int colIdx = 0; colIdx < W/2; ++colIdx)
    {
        // Fetch an src element pair from global memory:
        __half2 elemPair = srcData[elemOffset + colIdx];
        // Save the src element to local buffer:
        dataLocal[colIdx] = elemPair;
        // Take the maximum of both the upper and the lower value:
        float maxElemPair = fmaxf(__high2float(elemPair), __low2float(elemPair));
        // Take the maximum of the previous maximum element and the maximum of the element pair:
        maxElem = fmaxf(maxElem, maxElemPair);
    }

    // Store maximum of this row in shared memory
    s_rowMaximum[row] = maxElem;

    __syncthreads();

    // 1b) Reduce all maxima from previous step to a global channel maximum and store it in the respective variable in shared memory.
    if (row == 0)
    {
#pragma unroll
        for (int rowIdx = 0; rowIdx < H; ++rowIdx)
        {
            maxElem = fmaxf(maxElem, s_rowMaximum[rowIdx]);
        }
        s_channelMaximum = maxElem;
    }
    __half2 tmpSumHalf2 = __half2half2(0.0f);
    __syncthreads();

    

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // PHASE 2: Calculate the pixel-wise Softmax numerators in each channel, and compute the output keypoint probability. 
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
    // 2a) Within each channel, calculate the Softmax numerator of each pixel (subtract the maximum from it, multiply with beta, exponentiate).
    //     Store the result in local memory. Also accumulate the result to tmpSumHalf2.

    // Convert the previously found maximum to __half2:
    __half2 maxElemHalf2 = __float2half2_rn(s_channelMaximum);

    // Using one thread per row, loop over all columns:
#pragma unroll
    for (int colIdx = 0; colIdx < W/2; ++colIdx)
    {
        // Take an element pair from the local buffer:
        __half2 elemPair = dataLocal[colIdx];
        
        // Perform the following__half2 operation: f(x) = exp(beta * (x - channelMaximum))
        __half2 result = h2exp( __hmul2(__hsub2(elemPair, maxElemHalf2), betaHalf2) );

        // Accumulate the results (required for the probability results per keypoint/channel):
        tmpSumHalf2 = __hadd2(result, tmpSumHalf2);

        // Overwrite the buffer element with the result since we need it for the final Softargmax step: 
        dataLocal[colIdx] = result;
    }
    s_sumRows[row] = __hadd(__low2half(tmpSumHalf2), __high2half(tmpSumHalf2));
    
    __syncthreads();

    // 2b) Reduce all sums from the previous step to a single sum per channel (acting as the Softmax denominator), apply rcp, and store it in the first element of the shared row array.
    //     Also calculate the output value for channel/keypoint probability by averaging the sum.
    
    if (row == 0)
    {
        // We do not want to divide by 0, that's why we init this value with epsilon:
        __half tmpSumHalf = epsilon;
        __half channelElements = __float2half(float(HW));
        // Using one thread, loop over all rows:
#pragma unroll
        for (int rowIdx = 0; rowIdx < H; ++rowIdx)
        {
            // Accumulate all rows' aggregated values:
            tmpSumHalf = __hadd(tmpSumHalf, s_sumRows[rowIdx]);
        }
        dstDataProb[channel] = __hdiv(tmpSumHalf, channelElements);
        s_softmaxDenominatorRcp = __half2half2(hrcp(tmpSumHalf));
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // PHASE 3: Calculate the pixel-wise Softargmax values for each channel/keypoint.
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    
    // 3a) Calculate the Softargmax value for each pixel.

    // Use a local copy of the Softmax denominator:
    __half2 softmaxDenominatorRcpHalf2 = s_softmaxDenominatorRcp;
    
    // Convert this thread's row index into __half2:
    __half2 rowHalf2 = __float2half2_rn(float(row));
    // Initialize the row-wise sums for the X and Y outputs (no epsilon needed):
    __half2 sumRowX = __float2half2_rn(0.0f);
    __half2 sumRowY = __float2half2_rn(0.0f);

    // Using one thread per row, loop over all columns:
#pragma unroll
    for (int colIdx = 0; colIdx < W/2; ++colIdx)
    {
        // Grab an element pair from the local buffer, computed from the previous step:
        __half2 elemPair = dataLocal[colIdx];
        // Calculate the pixel-wise Softmax probability of the element pair within this channel:
        __half2 prob =  __hmul2(elemPair, softmaxDenominatorRcpHalf2);
        
        // Accumulate the result of f(prob, col) = prob * col and f(prob, row) = prob * col to the row-wise sums: 
        sumRowX = __hfma2(prob, make_half2(float(2*colIdx), float(2*colIdx+1)), sumRowX);
        sumRowY = __hfma2(prob, rowHalf2, sumRowY);
    }
    // Now sum-reduce the __half2 elements to a single element:
    s_sumRows[row] = __hadd(__low2half(sumRowX), __high2half(sumRowX));
    s_sumCols[row] = __hadd(__low2half(sumRowY), __high2half(sumRowY));
    __syncthreads();

    // 3b) Reduce all sums from the previous step to the output X and Y value for this channel:
    if (row == 0)
    {
        // Init the channel-wise sums for X and Y outputs:
        __half sumChannelX = __float2half(0.0f);
        __half sumChannelY = __float2half(0.0f);
        // Iterate over all sums in shared memory and accumulate them:
#pragma unroll
        for (int rowIdx = 0; rowIdx < H; ++rowIdx)
        {
            sumChannelX += s_sumRows[rowIdx];
            sumChannelY += s_sumCols[rowIdx];
        }
        // Store the outputs in global memory:
        dstDataX[channel] = sumChannelX;
        dstDataY[channel] = sumChannelY;
    }

}

template <int H, int W> __global__ void softargmax_kernel_float(const float *srcData, 
   float *dstDataX,
   float *dstDataY,
   float *dstDataProb,
   const int HW,
   const int CHW,
   float beta,
   float epsilon)
{

    // Shared memory for storing the maxima per row and after reduction per channel:
    __shared__ float s_rowMaximum[H];
    __shared__ float s_channelMaximum;
    // Shared memory for storing the intermediate sums per row and per column, and for saving the softmax denominator (as reciprocate):
    __shared__ float s_sumRows[H];
    __shared__ float s_sumCols[H];
    __shared__ float s_softmaxDenominatorRcp;

    // Instead of fetching input data from global memory twice, we will use a local buffer per thread:
    float dataLocal[W];

    // Information we can infer from this thread:
    const int batch = blockIdx.x;
    const int channel = blockIdx.y;
    const int row = threadIdx.x;

    // This is the starting index of the input elements in float strides
    const int elemOffset = (batch * CHW + channel * HW + row * W);

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // PHASE 1: Calculate the maximum per channel for "accurate" Softmax mode.
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // 1a) Find maximum per row:
    float maxElem = -FLT_MAX;
#pragma unroll
    for (int colIdx = 0; colIdx < W; ++colIdx)
    {
        // Fetch an src element pair from global memory:
        float elem = srcData[elemOffset + colIdx];
        // Save the src element to local buffer:
        dataLocal[colIdx] = elem;
        // Take the maximum of the previous maximum element and the current element:
        maxElem = fmaxf(maxElem, elem);

    }

    // Store maximum of this row in shared memory
    s_rowMaximum[row] = maxElem;

    __syncthreads();

    // 1b) Reduce all maxima from previous step to a global channel maximum and store it in the respective variable in shared memory.
    if (row == 0)
    {
#pragma unroll
        for (int rowIdx = 0; rowIdx < H; ++rowIdx)
        {
            maxElem = fmaxf(maxElem, s_rowMaximum[rowIdx]);
        }
        s_channelMaximum = maxElem;
    }
    float tmpSum = 0.0f;
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // PHASE 2: Calculate the pixel-wise Softmax numerators in each channel, and compute the output keypoint probability. 
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // 2a) Within each channel, calculate the Softmax numerator of each pixel (subtract the maximum from it, multiply with beta, exponentiate).
    //     Store the result in local memory. Also accumulate the result to tmpSum.
    maxElem = s_channelMaximum;
     // Using one thread per row, loop over all columns:
#pragma unroll
    for (int colIdx = 0; colIdx < W; ++colIdx)
    {
        // Take an element from the local buffer:
        float elem = dataLocal[colIdx];
        // Perform the following operation: f(x) = exp(beta * (x - channelMaximum))
        float result = expf((elem - maxElem) * beta) ;
        // Accumulate the results (required for the probability results per keypoint/channel):
        tmpSum += result;
        // Overwrite the buffer element with the result since we need it for the final Softargmax step: 
        dataLocal[colIdx] = result;

    }
    s_sumRows[row] = tmpSum;
    
    __syncthreads();

    // 2b) Reduce all sums from the previous step to a single sum per channel (acting as the Softmax denominator), apply rcp, and store it in the first element of the shared row array.
    //     Also calculate the output value for channel/keypoint probability by averaging the sum.

    if (row == 0)
    {
        // We do not want to divide by 0, that's why we init this value with epsilon:
        tmpSum = epsilon;
        // Using one thread, loop over all rows:
#pragma unroll
        for (int rowIdx = 0; rowIdx < H; ++rowIdx)
        {
            // Accumulate all rows' aggregated values:
            tmpSum += s_sumRows[rowIdx];
        }
        dstDataProb[channel] = tmpSum/float(HW);
        s_softmaxDenominatorRcp = 1.0f/tmpSum;

    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // PHASE 3: Calculate the pixel-wise Softargmax values for each channel/keypoint.
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // 3a) Calculate the Softargmax value for each pixel.

    // Use a local copy of the Softmax denominator:
    float softmaxDenominatorRcp = s_softmaxDenominatorRcp;

    // Initialize the row-wise sums for the X and Y outputs (no epsilon needed):
    float sumRowsX = 0.0f;
    float sumRowsY = 0.0f;
    float rowFloat = float(row);
    for (int colIdx = 0; colIdx < W; ++colIdx)
    {
        // Grab an element pair from the local buffer, computed from the previous step:
        float elem = dataLocal[colIdx];
        float prob =  elem * softmaxDenominatorRcp;

        // Calculate the pixel-wise Softmax probability of the element pair within this channel:
        sumRowsX = fmaf(prob, float(colIdx), sumRowsX);
        sumRowsY = fmaf(prob, rowFloat, sumRowsY);

    }
    s_sumRows[row] = sumRowsX;
    s_sumCols[row] = sumRowsY;
    __syncthreads();

    // 3b) Reduce all sums from the previous step to the output X and Y value for this channel:

    if (row == 0)
    {
        float sumX = 0.0f;
        float sumY = 0.0f;
         // Iterate over all sums in shared memory and accumulate them:
#pragma unroll
        for (int rowIdx = 0; rowIdx < H; ++rowIdx)
        {
            sumX += s_sumRows[rowIdx];
            sumY += s_sumCols[rowIdx];
        }
        // Store the outputs in global memory:
        dstDataX[channel] = sumX;
        dstDataY[channel] = sumY;
    }

}

void softargmax_kernel_fl(hipStream_t stream, const void* const* inputs, void **outputs,
    const int N, const int C, const int H, const int W, float beta, float epsilon, bool isfp16)
{    
    if (isfp16) {
        softargmax_kernel_half<g_H_Fl, g_W_Fl><<<dim3(N, C, 1), dim3(W), 0, stream>>>(
            reinterpret_cast<const __half2 *>(inputs[0]), reinterpret_cast<__half *>(outputs[0]),
            reinterpret_cast<__half *>(outputs[0]) + C, reinterpret_cast<__half *>(outputs[1]),
            H * W, C * H * W, __float2half(beta), __float2half(epsilon));
    } else {
        softargmax_kernel_float<g_H_Fl, g_W_Fl><<<dim3(N, C, 1), dim3(W), 0, stream>>>(
            reinterpret_cast<const float *>(inputs[0]), reinterpret_cast<float *>(outputs[0]),
            reinterpret_cast<float *>(outputs[0]) + C, reinterpret_cast<float *>(outputs[1]),
            H * W, C * H * W, beta, epsilon);
    }
}
